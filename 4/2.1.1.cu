#include <stdio.h>
#include <hip/hip_runtime.h>

// macros:
#define widthField 2
#define precisionField 0
struct Matrix;
__global__ void init_GPU (double *p, int rows, int cols);
__global__ void mul_GPU (double *m1, double *m2, double *p, int rows, int x, int cols);
struct Matrix
{
    int rows, cols;
    double *device_pointer, *host_pointer;
    int flag = 0;
    Matrix () : rows (0), cols (0), device_pointer (NULL), host_pointer (NULL)
    {
        return;
    }
    Matrix (int r, int c) : Matrix ()
    {
        rows = r;
        cols = c;
        alloc ();
        return;
    }
    Matrix (const Matrix &M)
    {
        printf ("\033[90mMatrix (const Matrix &M)\033[m\n");
        rows = M.rows;
        cols = M.cols;
        hipMalloc (&device_pointer, rows * cols * sizeof (double));
        hipMemcpy (device_pointer, M.device_pointer, rows * cols * sizeof (double), hipMemcpyDeviceToDevice);
        host_pointer = (double *) (malloc (rows * cols * sizeof (double)));
        memcpy (host_pointer, M.host_pointer, rows * cols * sizeof (double));
        return;
    }
    Matrix (Matrix &&M)
    {
        printf ("\033[90mMatrix (const Matrix &&M)\033[m\n");
        rows = M.rows;
        cols = M.cols;
        device_pointer = M.device_pointer;
        host_pointer = M.host_pointer;
        M.rows = M.cols = 0;
        M.device_pointer = M.host_pointer = NULL;
        // M.clear ();
        return;
    }
    Matrix operator = (Matrix &M)
    {
        printf ("\033[90mMatrix operator = (Matrix &M)\033[m\n");
        clear ();
        rows = M.rows;
        cols = M.cols;
        hipMalloc (&device_pointer, rows * cols * sizeof (double));
        hipMemcpy (device_pointer, M.device_pointer, rows * cols * sizeof (double), hipMemcpyDeviceToDevice);
        host_pointer = (double *) (malloc (rows * cols * sizeof (double)));
        memcpy (host_pointer, M.host_pointer, rows * cols * sizeof (double));
        return *this;
    }
    Matrix operator = (Matrix &&M)
    {
        printf ("\033[90mMatrix operator = (Matrix &&M)\033[m\n");
        rows = M.rows;
        cols = M.cols;
        device_pointer = M.device_pointer;
        host_pointer = M.host_pointer;
        M.rows = M.cols = 0;
        M.device_pointer = M.host_pointer = NULL;
        return *this;
    }
    ~Matrix ()
    {
        printf ("\033[90m~Matrix () : %p, %p\033[m\n", device_pointer, host_pointer);
        if (NULL != device_pointer)
        {
            hipFree (device_pointer);
        }
        if (NULL != host_pointer)
        {
            free (host_pointer);
        }
        rows = cols = 0;
        device_pointer = host_pointer = NULL;
        return;
    }
    void alloc ()
    {
        hipMalloc (&device_pointer, rows * cols * sizeof (double));
        host_pointer = (double *) (malloc (rows * cols * sizeof (double)));
        // printf ("hello");
        return;
    }
    void clear ()
    {
        // printf ("%p, %p\n", device_pointer, host_pointer);
        if (NULL != device_pointer)
        {
            hipFree (device_pointer);
        }
        if (NULL != host_pointer)
        {
            free (host_pointer);
        }
        rows = cols = 0;
        device_pointer = host_pointer = NULL;
        return;
    }
    // display works on host matrix
    void display ()
    {
        for (int i = 0; i < rows; i++)
        {
            for (int j = 0; j < cols; j++)
            {
                printf (" %*.*lf ", widthField, precisionField, host_pointer[i * cols + j]);
            }
            printf ("\n");
        }
        return;
    }
    void display (const Matrix &M)
    {
        if (cols == M.cols && rows == M.rows)
        {
            for (int i = 0; i < rows; i++)
            {
                // first matrix: 
                for (int j = 0; j < cols; j++)
                {
                    printf (" %*.*lf ", widthField, precisionField, host_pointer[i * cols + j]);
                }
                printf (" |  "); // seperator
                // second matrix: 
                for (int j = 0; j < cols; j++)
                {
                    printf (" %*.*lf ", widthField, precisionField, M.host_pointer[i * cols + j]);
                }
                printf ("\n");
            }
        }
        return;
    }
    void init ()
    {
        dim3 block (1, 1, 1);
        dim3 grid (rows, cols, 1);
        init_GPU <<<grid, block>>> (device_pointer, rows, cols);
        hipDeviceSynchronize ();
        // printf ("\033[31mhere\033[m");
        device2host ();
        // printf ("here");
        return;
    }
    void host2device ()
    {
        hipMemcpy (device_pointer, host_pointer, cols * rows * sizeof (double), hipMemcpyHostToDevice);
        return;
    }
    void device2host ()
    {
        hipMemcpy (host_pointer, device_pointer, cols * rows * sizeof (double), hipMemcpyDeviceToHost);
        return;
    }
    Matrix operator * (const Matrix &M)
    {
        if (cols != M.rows)
        {
            printf ("Matrix1 (%dX%d); Matrix2 (%dX%d)\n", rows, cols, M.rows, M.cols);
            return Matrix ();
        }
        Matrix p (rows, M.cols);
        dim3 block (1, 1, 1);
        dim3 grid (rows, M.cols, 1);
        mul_GPU <<< block, grid>>> (device_pointer, M.device_pointer, p.device_pointer, rows, cols, M.cols);
        hipDeviceSynchronize ();
        p.device2host ();
        p.display ();
        return p;
    }
};

__global__ void init_GPU (double *p, int rows, int cols)
{
    int r = threadIdx.x + blockIdx.x * blockDim.x; // x = rows
    int c = threadIdx.y + blockIdx.y * blockDim.y; // y = cols
    // printf ("%d;%d;%d;%d\n", r, c, M.rows, M.cols);
    if (r < rows && c < cols)
    {
        // printf ("<%d>", r * M.cols + c);
        p[r * cols + c] = ((double) (r * cols + c));
        // printf ("%lf ", M.device_pointer[r * M.cols + c]);
    }
    return;
}
__global__ void mul_GPU (double *m1, double *m2, double *p, int rows, int x, int cols)
{
    int Row = blockIdx.x * blockDim.x + threadIdx.x;
    int Col = blockIdx.y * blockDim.y + threadIdx.y;
    if (Row < rows && Col < cols)
    {
        // printf ("{%d,%d}", Row, Col);
        double a = 0;
        for (int k = 0; k < x; k++)
        {
            // printf ("(%.0f,%.0f)", m1[Row * cols + k], m2[k * rows + Col]);
            a += m1[Row * x + k] * m2[k * cols + Col];
        }
        p[Row * cols + Col] = a;
        // printf ("=<%f>\n", a);
    }
    return;
}
// void display_Matrix (Matrix M)
// {
//     int idx;
//     for (int i = 0; i < M.rows; i++)
//     {
//         for (int j = 0; j < M.cols; j++)
//         {
//             idx = i * M.cols + j;
//             printf (" %*.*lf ", widthField, precisionField, M.host_pointer[idx]);
//         }
//         printf ("\n");
//     }
// }
// __device__ int random_int ()
// {
//     static int i = 12345678;
//     i *= 0xf9f9f9f9, i++;
//     return i;
// }

// void initialize_Matrix (Matrix M)
// {
//     dim3 block (1, 1, 1);
//     dim3 grid (M.rows, M.cols, 1);
//     initialize_Matrix_GPU <<<grid, block>>> (M);
//     cudaDeviceSynchronize ();
//     return;
// }

// void allocate_Matrix (Matrix *m, int rows, int cols)
// {
//     m->rows = rows;
//     m->cols = cols;
//     cudaMalloc (&(m->device_pointer), rows * cols * sizeof (double));
//     m->host_pointer = (double *) malloc (rows * cols * sizeof (double));
//     return;
// }
int main ()
{
    // int Width = N;
    // int nx = Width;
    // int ny = Width;
    // int nxy = nx * ny;
    Matrix A (2, 8), B (8, 8);
    A.init ();
    A.display ();
    printf ("-----------------\n");
    B.init ();
    B.display ();
    printf ("-----------------\n");
    Matrix C = A * B;
    // C.display ();
    // C.display ();
    // allocate_Matrix (&A, 4, 8);
    // transfer_Matrix_h2d (A);
    // initialize_Matrix (A);
    // transfer_Matrix_d2h (A);
    // display_Matrix (A);

    
    // allocate_Matrix (&B, 8, 2);
    // allocate_Matrix (&C, 2, 4);
    
    // initialize_Matrix <<<

    // int nBytes = nxy * sizeof (float);
    // printf ("Matrix size: nx %d ny %d\n", nx, ny);
    
    // float *h_A, *h_B, *h_C;
    // h_A = (float *) (malloc (nBytes));
    // h_B = (float *) malloc (nBytes);
    // h_C = (float *) malloc (nBytes);
    
    // initialData (h_A, nxy);
    // initialData (h_B, nxy);
    
    // float *d_MatA, *d_MatB, *d_MatC;
    // cudaMalloc ((void **) &d_MatA, nBytes);
    // cudaMalloc ((void **) &d_MatB, nBytes);
    // cudaMalloc ((void **) &d_MatC, nBytes);


    // cudaMemcpy ((void *) d_MatA, h_A, nBytes, cudaMemcpyHostToDevice);
    // cudaMemcpy ((void *) d_MatB, h_B, nBytes, cudaMemcpyHostToDevice);

    // int bdimx = 16;
    // int bdimy = 16;

    // dim3 block (bdimx, bdimy, 1);
    // dim3 grid ((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y, 1);

    // MatrixMulKernel <<<grid, block>>> (d_MatA, d_MatB, d_MatC, Width);
    // cudaDeviceSynchronize ();

    // cudaMemcpy (h_C, d_MatC, nBytes, cudaMemcpyDeviceToHost);

    // printf ("Matrix A is=\n");
    // displayMatrix (h_A, nx, ny, 2);
    // printf ("Matrix B is=\n");
    // displayMatrix (h_B, nx, ny, 2);
    // printf ("The Product of Matrix A and Matrix B is=\n");
    // displayMatrix (h_C, nx, ny, 5);

    // cudaFree (d_MatA);
    // cudaFree (d_MatB);
    // cudaFree (d_MatC);

    // free (h_A);
    // free (h_B);
    // free (h_C);

    // cudaDeviceReset ();

    return 0;

}