#include <stdio.h>
#include "matrix.cuh"
int main ()
{
    srand (time (NULL));
    Matrix A (4, 3);
    A.init ();
    Matrix AT = ~A;
    printf ("Matrix A:\n");
    A.display ();
    printf ("Matrix AT:\n");
    AT.display ();
    Matrix P = A * AT;
    printf ("Matrix P:\n");
    P.display ();
    // Matrix D = C * C;
    // printf("Matrix D:\n");
    // D.display();
    // C + 
    // Matrix TA = ~A, TB = ~B;
    // printf ("\033[4;31mMatrix A:\033[m\n");
    // A.display ();
    // printf ("\033[4;31mMatrix B:\033[m\n");
    // B.display ();
    // printf ("\033[4;31mMatrix TA:\033[m\n");
    // TA.display ();
    // printf ("\033[4;31mMatrix TB:\033[m\n");
    // TB.display ();
    // Matrix PAB = A * B;
    // Matrix PTATB = TA * TB;
    // printf ("\033[4;31mMatrix PAB:\033[m\n");
    // PAB.display ();
    // printf ("\033[4;31mMatrix PTATB:\033[m\n");
    // PTATB.display ();
    // Matrix D = PAB - PTATB;
    // printf ("\033[4;31mMatrix D:\033[m\n");
    // D.display ();
    
    hipDeviceReset ();
    return 0;
}